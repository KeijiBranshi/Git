#include "hip/hip_runtime.h"
#include "qmd.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

int DEBUG;

int main(int argc, char* argv[]) {
  int step;

  MPI_Init(&argc,&argv);
  MPI_Comm_size(MPI_COMM_WORLD, &nProc);
  MPI_Comm_rank(MPI_COMM_WORLD, &myid);
  hipSetDevice(myid%2);
  usingCUDA = true;
  DEBUG = false;

  init_Parameters();
  init_Variables();
  init_Propogators();
  init_WaveFunction();
  if (usingCUDA) init_GPU();

  for (step=1; step<=nStep; ++step) {
    single_Step();
    if (step%energyInterval == 0) {
      calculate_Energy();
      if (myid==0) printf("%le %le %le %le\n", deltaTime*step, expected[_T_], expected[_V_], expected[_E_]);
      //if (myid == 0) printf("%le, %le\n", deltaTime*step, expected[_V_]);
    }
  }

  cleanUpVariables();
  if (usingCUDA) GPU_Finalize();
  MPI_Finalize();
  return 0;
}

/* Initialize GPU communication and variables **********************************************/
void init_GPU() {
  size_t size = 2*(MESH_SIZE)*sizeof(double);

  hipMalloc((double**) &dev_psi, size);
  hipMalloc((double**) &dev_workPsi, size);
  hipMalloc((double**) &dev_TDiag_half, 2 * 2 * sizeof(double));
  hipMalloc((double**) &dev_TDiag_full, 2 * 2 * sizeof(double));
  hipMalloc((double**) &dev_TUpper_half, size);
  hipMalloc((double**) &dev_TUpper_full, size);
  hipMalloc((double**) &dev_TLower_half, size);
  hipMalloc((double**) &dev_TLower_full, size);
  hipMalloc((double**) &dev_VPropogator, size);

  hostToDevice(&TDiag[FULL], dev_TDiag_full, 1);
  hostToDevice(TUpper[FULL], dev_TUpper_full, MESH_SIZE);
  hostToDevice(TLower[FULL], dev_TLower_full, MESH_SIZE);

  hostToDevice(&TDiag[HALF], dev_TDiag_half, 1);
  hostToDevice(TUpper[HALF], dev_TUpper_half, MESH_SIZE);
  hostToDevice(TLower[HALF], dev_TLower_half, MESH_SIZE);
}

/* Cleans up cudaMalloced memory */
void GPU_Finalize() {
  hipFree(&dev_psi);
  hipFree(&dev_workPsi);
  hipFree(&dev_TDiag_half);
  hipFree(&dev_TDiag_full);
  hipFree(&dev_TUpper_half);
  hipFree(&dev_TUpper_full);
  hipFree(&dev_TLower_half);
  hipFree(&dev_TLower_full);
  hipFree(&dev_VPropogator);
}

/* Initialize parameters for the simulation by reading in from a file **********************/
void init_Parameters() {
  scanf("%le", &Lx);
  scanf("%le", &deltaTime);
  scanf("%d", &nStep);
  scanf("%d", &energyInterval);
  scanf("%le %le %le", &x0, &s0, &e0);
  scanf("%le %le", &barrierHeight, &barrierWidth);
  scanf("%le", &edgePotential);

  if (DEBUG) {
    printf("Lx = %le\n", Lx);
    printf("deltaT = %le\n", deltaTime);
    printf("nStep = %d\n", nStep);
    printf("energyInterval = %d\n", energyInterval);
    printf("x0, s0, e0 = %le %le %le\n", x0, s0, e0);
    printf("bH, BW = %le %le\n", barrierHeight, barrierWidth);
    printf("edgePotential = %le\n", edgePotential);
  }

  MPI_Bcast(&Lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&deltaTime, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&nStep, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&energyInterval, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&x0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&s0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&e0, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&barrierWidth, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&barrierHeight, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(&edgePotential, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  // Mesh Size
  dx = (double) Lx / NX;
}

void init_Variables() {
  int i,j;

  host_psi = (double**) malloc(MESH_SIZE * sizeof(double*));
  for (i=0; i<MESH_SIZE; ++i) {
    host_psi[i] = (double *) malloc(sizeof(double)*2);
  }

  workPsi = (double**) malloc(MESH_SIZE * sizeof(double*));
  for (i=0; i<MESH_SIZE; ++i) {
    workPsi[i] = (double *) malloc (2 * sizeof(double));
  }

  TDiag = (double**) malloc(sizeof(double*) * 2);
  for (i=0; i<2; ++i) {
    TDiag[i] = (double*) malloc(sizeof(double) * 2);
  }

  TUpper = (double***) malloc(sizeof(double**) * 2);
  for (i=0; i<2; ++i) {
    TUpper[i] = (double**) malloc(sizeof(double*) * MESH_SIZE);
    for (j=0; j<MESH_SIZE; ++j) {
      TUpper[i][j] = (double*) malloc(sizeof(double) * COMPLEX);
    }
  }

  TLower = (double***) malloc(sizeof(double**) * 2);
  for (i=0; i<2; ++i) {
    TLower[i] = (double**) malloc(sizeof(double*) * MESH_SIZE);
    for (j=0; j<MESH_SIZE; ++j) {
      TLower[i][j] = (double*) malloc(sizeof(double) * COMPLEX);
    }
  }

  VPropogator = (double**) malloc(sizeof(double*) * MESH_SIZE);
  for (i=0; i<MESH_SIZE; ++i) {
    VPropogator[i] = (double *) malloc(sizeof(double)*2);
  }

  potential = (double*) malloc(sizeof(double) * MESH_SIZE);
}

void init_Propogators() {
  double a, exp_p[2], ePlus[2], eMinus[2];
  int i,c,upper,lower,step;                 //for iterating
  double x;

  a = 0.5 / (dx * dx);                      //diagonal element for kinetic propogator.

  // Construct the Kinetic Propogators
  for (step=0; step<2; ++step) {
    // Get espilon(1|2)(+|-) values
    exp_p[0] = cos(-(step+1) * deltaTime * a);
    exp_p[1] = sin(-(step+1) * deltaTime * a);
    ePlus[0] = 0.5 * (1.0 + exp_p[0]);
    ePlus[1] = 0.5 * exp_p[1];
    eMinus[0] = 0.5 * (1.0 - exp_p[0]);
    eMinus[1] = -0.5 * exp_p[1];

    for (c=0; c<2; ++c) {
      TDiag[step][c] = ePlus[c];
    }

    for (i=1; i<=NX; ++i) {
      if (step == HALF) {
        upper = i%2;
        lower = (i+1)%2;
      } else {
        upper = (i+1)%2;
        lower = i%2;
      }
      for (c=0; c<2; ++c) {
        //if (myid==0) printf("Loop: step=%i i=%i, c=%i\n", step, i, c);
        TUpper[step][i][c] = upper * eMinus[c];
        TLower[step][i][c] = lower * eMinus[c];
      }
    }
  }

  // Construct Potential Propogators
  for (i=1; i<=NX; ++i) {
    x = (dx*i + Lx*myid);

    // Edge Potential
    if ((myid==0 && i==1) || (myid==nProc-1 && i==NX)) {
      potential[i] = edgePotential;
    } else if (0.5*(Lx*nProc-barrierWidth)<x && x<0.5*(Lx*nProc+barrierWidth)){
      potential[i] = barrierHeight;
    } else
      potential[i] = 0;

    VPropogator[i][RE] = cos(-0.5 * deltaTime * potential[i]);
    VPropogator[i][IM] = sin(-0.5 * deltaTime * potential[i]);
  }
}

void init_WaveFunction() {
  int sx, c;
  double x, gaussian, normalize;
  double psiSquared, temp;
  double DXSX, LXMYID;          //for debugging

  // Calculuate Psi point-by-point
  for (sx=1; sx<=NX; ++sx) {
    DXSX = (double) dx * sx;
    LXMYID = (double) Lx * myid;
    temp = DXSX + LXMYID;

    x = temp - x0;
    gaussian = exp((-0.25 * x * x) / (s0 * s0));
    host_psi[sx][RE] = gaussian * cos(sqrt(2.0 * e0) * x);
    host_psi[sx][IM] = gaussian * sin(sqrt(2.0 * e0) * x);
  }

  // Normalize
  temp = 0.0;
  for (sx=1; sx<=NX; ++sx) {
    for (c=0; c<2; ++c) {
      temp += (host_psi[sx][c]*host_psi[sx][c]);
    }
  }
  MPI_Allreduce(&temp, &psiSquared, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);
  psiSquared *= dx;
  //printf("<Proc: %i> PSI SQUARED: %le\n", myid, psiSquared);

  normalize = 1.0 / sqrt(psiSquared);
  for (sx=1; sx<=NX; ++sx) {
    for (c=0; c<2; ++c) {
      host_psi[sx][c] *= normalize;
    }
  }
}

void cleanUpVariables() {
  int i,j;

  for (i=0; i<MESH_SIZE; ++i) {
    free(host_psi[i]);
  }
  free(host_psi);

  for (i=0; i<MESH_SIZE; ++i) {
    free(workPsi[i]);
  }
  free(workPsi);

  for (i=0; i<2; ++i) {
    free(TDiag[i]);
  }
  free(TDiag);

  for (i=0; i<2; ++i) {
    for (j=0; j<MESH_SIZE; ++j) {
      free(TUpper[i][j]);
    }
    free(TUpper[i]);
  }
  free(TUpper);

  for (i=0; i<2; ++i) {
    for (j=0; j<MESH_SIZE; ++j) {
      free(TLower[i][j]);
    }
    free(TLower[i]);
  }
  free(TLower);

  for (i=0; i<MESH_SIZE; ++i) {
    free(VPropogator[i]);
  }
  free(VPropogator);

  free(potential);
}


/**************************************************************************************
 * Time Stepping/Propogation Functions
 **************************************************************************************/

/* SIngle time step in Quantum Dynamics simulation */
void single_Step() {
  potential_Propogation();          //Half Potential Propogator

  kinetic_Propogation(HALF);        //Half Kinetic Propogator
  kinetic_Propogation(FULL);        //Full Kinetic Propogator
  kinetic_Propogation(HALF);        //Half Kinetic Propogator

  potential_Propogation();

  //getPsiSquared();
}

void getPsiSquared() {
  double temp, psiSquared;
  int sx, c;

  temp = 0.0;
  psiSquared = 0.0;
  for (sx=1; sx<=NX; ++sx) {
    for (c=0; c<2; ++c) {
      temp += (host_psi[sx][c]*host_psi[sx][c]);
    }
  }
  MPI_Allreduce(&temp, &psiSquared, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);
  psiSquared *= dx;

  if (myid==0) printf("Psi Squared: %le\n", psiSquared);
}

/* Half Potential Energy propogator Function */
void potential_Propogation() {
  if (usingCUDA) {
    hostToDevice(host_psi, dev_psi, MESH_SIZE);
    gpu_Potential_Prop <<<gridDim, blockDim>>> (dev_psi, dev_VPropogator);
    deviceToHost(dev_psi, host_psi, MESH_SIZE);
  }
  else {
    regular_Potential_Prop();
  }
}

__global__ void gpu_Potential_Prop(double *psi, double *vProp) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int sx, s_Re, s_Im;
  double workPsi_Re, workPsi_Im;
  sx= tid+1;
  s_Re = 2*sx;
  s_Im = 2*sx+1;

  workPsi_Re = vProp[s_Re]*psi[s_Re] - vProp[s_Im]*psi[s_Im];
  workPsi_Im = vProp[s_Re]*psi[s_Im] + vProp[s_Im]*psi[s_Re];
  psi[s_Re] = workPsi_Re;
  psi[s_Im] = workPsi_Im;
}

void regular_Potential_Prop() {
  int sx;
  double workPsi_Re, workPsi_Im;

  for (sx=1; sx<=NX; ++sx) {
    workPsi_Re = VPropogator[sx][RE]*host_psi[sx][RE] - VPropogator[sx][IM]*host_psi[sx][IM];
    workPsi_Im = VPropogator[sx][RE]*host_psi[sx][IM] + VPropogator[sx][IM]*host_psi[sx][RE];
    host_psi[sx][RE]= workPsi_Re;
    host_psi[sx][IM]= workPsi_Im;
  }
}

/* Half|Full Kinetic Energy Propogator Function */
void kinetic_Propogation(int stepSize) {
  //First apply periodic bounds
  periodic_Bounds();

  if (usingCUDA) {
    hostToDevice(host_psi, dev_psi, MESH_SIZE);

    if (stepSize == FULL) {
      gpu_Kinetic_Prop<<<gridDim, blockDim>>>(dev_psi, dev_workPsi, dev_TDiag_full, dev_TLower_full, dev_TUpper_full);
    }
    else {
      gpu_Kinetic_Prop<<<gridDim, blockDim>>>(dev_psi, dev_workPsi, dev_TDiag_half, dev_TLower_half, dev_TUpper_half);
    }

    gpu_WorkToPsi<<<gridDim, blockDim>>>(dev_workPsi, dev_psi);
    deviceToHost(dev_psi, host_psi, MESH_SIZE);
  }
  else {
    regular_Kinetic_Prop(stepSize);
  }
}

__global__ void gpu_Kinetic_Prop(double *psi, double *work, double *al, double *blx, double *bux) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int sx, s_Re, s_Im, l_Re, l_Im, u_Re, u_Im;
  double workPsi_Re, workPsi_Im;
  sx= tid+1;
  s_Re = 2*sx;
  s_Im = 2*sx+1;
  l_Re = 2*(sx-1);
  l_Im = 2*(sx-1)+1;
  u_Re = 2*(sx+1);
  u_Im = 2*(sx+1)+1;

  workPsi_Re = al[0]*psi[s_Re] - al[1]*psi[s_Im];
  workPsi_Im = al[0]*psi[s_Im] + al[1]*psi[s_Re];
  workPsi_Re += blx[s_Re]*psi[l_Re] - blx[s_Im]*psi[l_Im];
  workPsi_Im += blx[s_Re]*psi[l_Im] + blx[s_Im]*psi[l_Re];
  workPsi_Re += bux[s_Re]*psi[u_Re] - bux[s_Im]*psi[u_Im];
  workPsi_Im += bux[s_Re]*psi[u_Im] + bux[s_Im]*psi[u_Re];

  work[s_Re] = workPsi_Re;
  work[s_Im] = workPsi_Im;
}

void regular_Kinetic_Prop(int t) {
  int sx, c;
  double workPsi_Re, workPsi_Im;

  for (sx=1; sx<=NX; ++sx) {
    workPsi_Re = TDiag[t][RE]*host_psi[sx][RE] - TDiag[t][IM]*host_psi[sx][IM];
    workPsi_Im = TDiag[t][RE]*host_psi[sx][IM] + TDiag[t][IM]*host_psi[sx][RE];
    workPsi_Re += (TLower[t][sx][RE]*host_psi[sx-1][RE]) - (TLower[t][sx][IM]*host_psi[sx-1][IM]);
    workPsi_Im += (TLower[t][sx][RE]*host_psi[sx-1][IM]) + (TLower[t][sx][IM]*host_psi[sx-1][RE]);
    workPsi_Re += (TUpper[t][sx][RE]*host_psi[sx+1][RE]) - (TUpper[t][sx][IM]*host_psi[sx+1][IM]);
    workPsi_Im += (TUpper[t][sx][RE]*host_psi[sx+1][IM]) + (TUpper[t][sx][IM]*host_psi[sx+1][RE]);

    workPsi[sx][RE] = workPsi_Re;
    workPsi[sx][IM] = workPsi_Im;
  }

  for (sx=1; sx<=NX; ++sx) {
    for (c=0; c<2; ++c) {
      host_psi[sx][c] = workPsi[sx][c];
    }
  }
}

/*************************************************************************************
 * HOST-DEVICE CONVERSION Functions
 ************************************************************************************/

__global__ void gpu_WorkToPsi(double *work, double *psi) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int sx, s_Re, s_Im;

  sx= tid+1;
  s_Re = 2*sx;
  s_Im = 2*sx+1;
  psi[s_Re] = work[s_Re];
  psi[s_Im] = work[s_Im];
}

void hostToDevice(double** host, double* device, int size) {
  int i,j;
  double *hostBuf;
  hostBuf = (double*) malloc(sizeof(double) * size * 2);

  for (i=0; i<size; ++i) {
    for (j=0; j<2; ++j) {
      hostBuf[2*i + j] = host[i][j];
    }
  }

  hipMemcpy((void*) device, hostBuf, 2*size*sizeof(double), hipMemcpyHostToDevice);
  free(hostBuf);
}

void deviceToHost(double* device, double** host, int size) {
  int i,j;
  double *devBuf;
  devBuf = (double*) malloc(sizeof(double) * size * 2);

  hipMemcpy((void*) devBuf, device, 2*size*sizeof(double), hipMemcpyDeviceToHost);
  for (i=0; i<size; ++i) {
    for (j=0; j<2; ++j) {
      host[i][j] = devBuf[2*i + j];
    }
  }
  free(devBuf);
}

/**************************************************************************************
 * MPI Related Functions
 *************************************************************************************/

/* Applies Periodic boundary conditions */
void periodic_Bounds() {
  int neighbor[2];
  double sendBuf[2], recvBuf[2];

  neighbor[right] = (myid + 1) % nProc;
  neighbor[left] = (myid - 1 + nProc) % nProc;

  // Send right, receive left
  sendBuf[0] = host_psi[NX][RE];
  sendBuf[1] = host_psi[NX][IM];
  MPI_Irecv(recvBuf, 2, MPI_DOUBLE, neighbor[left], DAEMON, MPI_COMM_WORLD, &request);
  MPI_Send(sendBuf, 2, MPI_DOUBLE, neighbor[right], DAEMON, MPI_COMM_WORLD);
  MPI_Wait(&request, &status);
  host_psi[0][RE] = recvBuf[RE];
  host_psi[0][IM] = recvBuf[IM];

  // Send left, receive right
  sendBuf[0] = host_psi[1][RE];
  sendBuf[1] = host_psi[1][IM];
  MPI_Irecv(recvBuf, 2, MPI_DOUBLE, neighbor[right], DAEMON+1, MPI_COMM_WORLD, &request);
  MPI_Send(sendBuf, 2, MPI_DOUBLE, neighbor[left], DAEMON+1, MPI_COMM_WORLD);
  MPI_Wait(&request, &status);
  host_psi[NX+1][RE] = recvBuf[RE];
  host_psi[NX+1][IM] = recvBuf[IM];
}

/* Total Energy Calculation at the current time step */
void calculate_Energy() {
  int sx, c;
  double a, b;

  // Apply periodic boundary conditions
  periodic_Bounds();

  // Tridiagonal KE operators
  a = 1.0 / (dx * dx);
  b = -0.5 / (dx * dx);

  // | work > = (-1/2) Laplacian | Psi >
  for (sx=1; sx<=NX; ++sx)
    for (c=0; c<2; ++c)
      workPsi[sx][c] = a*host_psi[sx][c] + b*(host_psi[sx-1][c] + host_psi[sx+1][c]);

  // Expected Value for Kinetic Energy
  expected[_T_] = 0.0;
  for (sx=1; sx<=NX; ++sx)
    expected[_T_] += (host_psi[sx][RE]*workPsi[sx][RE] + host_psi[sx][IM]*workPsi[sx][IM]);
  expected[_T_] *= dx;
  MPI_Allreduce(&expected[_T_], &expected[_T_], 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);

  // Expected Value for Potential Energy
  expected[_V_] = 0.0;
  for (sx=1; sx<=NX; ++sx)
    expected[_V_] += potential[sx]*(host_psi[sx][RE]*host_psi[sx][RE] + host_psi[sx][IM]*host_psi[sx][IM]);
  expected[_V_] *= dx;
  MPI_Allreduce(&expected[_V_], &expected[_V_], 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);

  // Expected total Energy
  expected[_E_] = expected[_T_] + expected[_V_];
}
